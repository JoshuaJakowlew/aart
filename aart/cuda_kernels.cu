#include "hip/hip_runtime.h"
#include <hip/hip_runtime.h>
#include <hip/hip_runtime.h>

#include "cuda_kernels.h"

__global__ void vectorAdd(const float* A, const float* B, float* C, int numElements) {
    int i = threadIdx.x;
    
    C[i] = A[i] + B[i];
}

void add_with_cuda(const float* A, const float* B, float* C, int numElements)
{
    float *gpuA, *gpuB, *gpuC;
    hipMalloc(&gpuA, sizeof(float) * numElements);
    hipMalloc(&gpuB, sizeof(float) * numElements);
    hipMalloc(&gpuC, sizeof(float) * numElements);

    hipMemcpy(gpuA, A, sizeof(float) * numElements, hipMemcpyHostToDevice);
    hipMemcpy(gpuB, B, sizeof(float) * numElements, hipMemcpyHostToDevice);
    hipMemcpy(gpuC, C, sizeof(float) * numElements, hipMemcpyHostToDevice);

    vectorAdd<<<1, numElements>>>(gpuA, gpuB, gpuC, numElements);

    hipMemcpy(C, gpuC, sizeof(float) * numElements, hipMemcpyDeviceToHost);
}